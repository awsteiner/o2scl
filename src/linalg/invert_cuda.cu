/*
  ───────────────────────────────────────────────────────────────────

  Copyright (C) 2025, Andrew W. Steiner

  This file is part of O2scl.

  O2scl is free software; you can redistribute it and/or modify
  it under the terms of the GNU General Public License as published by
  the Free Software Foundation; either version 3 of the License, or
  (at your option) any later version.

  O2scl is distributed in the hope that it will be useful,
  but WITHOUT ANY WARRANTY; without even the implied warranty of
  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
  GNU General Public License for more details.

  You should have received a copy of the GNU General Public License
  along with O2scl. If not, see <http://www.gnu.org/licenses/>.

  ───────────────────────────────────────────────────────────────────
*/
#include "invert_cuda.h"

#include <hip/hip_runtime.h>
#include <hipsolver.h>

using namespace o2scl_linalg;

int o2scl_linalg::cholesky_decomp_cuda(const size_t n, std::vector<double> &A) {

  // Note that the function hipsolverDnDpotrf presumes that the matrix
  // is stored in column-major, rather than row-major order. This
  // isn't a problem for this function, however, because the matrix
  // is assumed to be symmetric. 
  
  // Allocate device memory
  double *d_A=0;
  hipError_t cudaStat=hipMalloc((void**)&d_A,n*n*sizeof(double));
  if (cudaStat != hipSuccess) {
    return 1;
  }
  
  // Copy data to device
  cudaStat=hipMemcpy(d_A,A.data(),n*n*sizeof(double),
                      hipMemcpyHostToDevice);
  if (cudaStat != hipSuccess) {
    hipFree(d_A);
    return 2;
  }
  
  // Create cuSolver handle
  hipsolverHandle_t cusolverH=0;
  hipsolverStatus_t cusolver_status=hipsolverDnCreate(&cusolverH);
  if (cusolver_status != HIPSOLVER_STATUS_SUCCESS) {
    hipFree(d_A);
    return 3;
  }
  
  // Get buffer size for potrf (Cholesky)
  int work_size=0;
  cusolver_status=hipsolverDnDpotrf_bufferSize
    (cusolverH,HIPBLAS_FILL_MODE_LOWER,n,d_A,n,&work_size);
  
  if (cusolver_status != HIPSOLVER_STATUS_SUCCESS) {
    hipsolverDnDestroy(cusolverH);
    hipFree(d_A);
    return 4;
  }
  
  // Allocate workspace and devInfo
  double *d_work=0;
  int *devInfo=0;
  cudaStat=hipMalloc((void**)&d_work,work_size*sizeof(double));
  if (cudaStat != hipSuccess) {
    hipsolverDnDestroy(cusolverH);
    hipFree(d_A);
    return 5;
  }
  
  cudaStat=hipMalloc((void**)&devInfo,sizeof(int));
  if (cudaStat != hipSuccess) {
    hipFree(d_work);
    hipsolverDnDestroy(cusolverH);
    hipFree(d_A);
    return 6;
  }
  
  // Cholesky decomposition (A=L*L^T)
  cusolver_status=hipsolverDnDpotrf
    (cusolverH,HIPBLAS_FILL_MODE_LOWER,n,d_A,n,d_work,
     work_size,devInfo);
  
  if (cusolver_status != HIPSOLVER_STATUS_SUCCESS) {
    hipFree(d_work);
    hipFree(devInfo);
    hipsolverDnDestroy(cusolverH);
    hipFree(d_A);
    return 7;
  }
  
  // Copy result back to host
  cudaStat=hipMemcpy(A.data(),d_A,n*n*sizeof(double),
                      hipMemcpyDeviceToHost);
  if (cudaStat != hipSuccess) {
    return 8;
  }
  
  return 0;
}

int matrix_invert_det_cholesky_cuda::invert
(size_t n, const std::vector<double> &A,
 std::vector<double> &A_inv) {

  // Make a copy of the original matrix since the original
  // will be destroyed?
  std::vector<double> Acopy(n*n);
  for(size_t i=0;i<n*n;i++) {
    Acopy[i]=A[i];
  }
  
  // Allocate device memory
  double *d_A=0;
  hipError_t cudaStat=hipMalloc((void**)&d_A,n*n*sizeof(double));
  if (cudaStat != hipSuccess) {
    return 1;
  }
  
  // Copy data to device
  cudaStat=hipMemcpy(d_A,Acopy.data(),n*n*sizeof(double),
                      hipMemcpyHostToDevice);
  if (cudaStat != hipSuccess) {
    hipFree(d_A);
    return 2;
  }
  
  // Create cuSolver handle
  hipsolverHandle_t cusolverH=0;
  hipsolverStatus_t cusolver_status=hipsolverDnCreate(&cusolverH);
  if (cusolver_status != HIPSOLVER_STATUS_SUCCESS) {
    hipFree(d_A);
    return 3;
  }
  
  // Get buffer size for potrf (Cholesky)
  int work_size=0;
  cusolver_status=hipsolverDnDpotrf_bufferSize
    (cusolverH,HIPBLAS_FILL_MODE_LOWER,n,d_A,n,&work_size);
  
  if (cusolver_status != HIPSOLVER_STATUS_SUCCESS) {
    hipsolverDnDestroy(cusolverH);
    hipFree(d_A);
    return 4;
  }
  
  // Allocate workspace and devInfo
  double *d_work=0;
  int *devInfo=0;
  cudaStat=hipMalloc((void**)&d_work,work_size*sizeof(double));
  if (cudaStat != hipSuccess) {
    hipsolverDnDestroy(cusolverH);
    hipFree(d_A);
    return 5;
  }
  
  cudaStat=hipMalloc((void**)&devInfo,sizeof(int));
  if (cudaStat != hipSuccess) {
    hipFree(d_work);
    hipsolverDnDestroy(cusolverH);
    hipFree(d_A);
    return 6;
  }
  
  // Cholesky decomposition (A=L*L^T)
  cusolver_status=hipsolverDnDpotrf
    (cusolverH,HIPBLAS_FILL_MODE_LOWER,n,d_A,n,d_work,
     work_size,devInfo);
  
  if (cusolver_status != HIPSOLVER_STATUS_SUCCESS) {
    hipFree(d_work);
    hipFree(devInfo);
    hipsolverDnDestroy(cusolverH);
    hipFree(d_A);
    return 7;
  }
  
  // Invert using Cholesky result
  cusolver_status=hipsolverDnDpotri
    (cusolverH,HIPBLAS_FILL_MODE_LOWER,n,d_A,n,d_work,
     work_size,devInfo);
  
  if (cusolver_status != HIPSOLVER_STATUS_SUCCESS) {
    hipFree(d_work);
    hipFree(devInfo);
    hipsolverDnDestroy(cusolverH);
    hipFree(d_A);
    return 8;
  }
  
  // Copy result back to host
  A_inv.resize(n*n);
  cudaStat=hipMemcpy(A_inv.data(),d_A,n*n*sizeof(double),
                      hipMemcpyDeviceToHost);
  if (cudaStat != hipSuccess) {
    return 9;
  }
  
  // Symmetrize the result (only lower triangle is filled)
  for (int i=0;i<n;++i) {
    for (int j=i+1;j<n;++j) {
      A_inv[i*n+j]=A_inv[j*n+i];
    }
  }
  
  // Clean up
  hipFree(d_A);
  hipFree(d_work);
  hipFree(devInfo);
  hipsolverDnDestroy(cusolverH);
  
  return 0;
}

int matrix_invert_det_cholesky_cuda::invert_det
(size_t n, const std::vector<double> &A,
 std::vector<double> &A_inv, double &A_det) {
  
  // Allocate device memory
  double *d_A=0;
  hipError_t cudaStat=hipMalloc((void**)&d_A,n*n*sizeof(double));
  if (cudaStat != hipSuccess) {
    return 1;
  }
  
  // Copy data to device
  cudaStat=hipMemcpy(d_A,A.data(),n*n*sizeof(double),
                      hipMemcpyHostToDevice);
  if (cudaStat != hipSuccess) {
    hipFree(d_A);
    return 2;
  }
  
  // Create cuSolver handle
  hipsolverHandle_t cusolverH=0;
  hipsolverStatus_t cusolver_status=hipsolverDnCreate(&cusolverH);
  if (cusolver_status != HIPSOLVER_STATUS_SUCCESS) {
    hipFree(d_A);
    return 3;
  }
  
  // Get buffer size for potrf (Cholesky)
  int work_size=0;
  cusolver_status=hipsolverDnDpotrf_bufferSize
    (cusolverH,HIPBLAS_FILL_MODE_LOWER,n,d_A,n,&work_size);
  
  if (cusolver_status != HIPSOLVER_STATUS_SUCCESS) {
    hipsolverDnDestroy(cusolverH);
    hipFree(d_A);
    return 4;
  }
  
  // Allocate workspace and devInfo
  double *d_work=0;
  int *devInfo=0;
  cudaStat=hipMalloc((void**)&d_work,work_size*sizeof(double));
  if (cudaStat != hipSuccess) {
    hipsolverDnDestroy(cusolverH);
    hipFree(d_A);
    return 5;
  }
  
  cudaStat=hipMalloc((void**)&devInfo,sizeof(int));
  if (cudaStat != hipSuccess) {
    hipFree(d_work);
    hipsolverDnDestroy(cusolverH);
    hipFree(d_A);
    return 6;
  }
  
  // Cholesky decomposition (A=L*L^T)
  cusolver_status=hipsolverDnDpotrf
    (cusolverH,HIPBLAS_FILL_MODE_LOWER,n,d_A,n,d_work,
     work_size,devInfo);
  
  if (cusolver_status != HIPSOLVER_STATUS_SUCCESS) {
    hipFree(d_work);
    hipFree(devInfo);
    hipsolverDnDestroy(cusolverH);
    hipFree(d_A);
    return 7;
  }

  // Copy Cholesky decomposition back to host to compute
  // determinant
  std::vector<double> chol(n*n);
  cudaStat=hipMemcpy(chol.data(),d_A,n*n*sizeof(double),
                      hipMemcpyDeviceToHost);

  // Compute determinant
  double sqrt_det=1.0;
  for(size_t i=0;i<n;i++) sqrt_det*=chol[i*n+i];
  A_det=sqrt_det*sqrt_det;
  
  // Invert using Cholesky result
  cusolver_status=hipsolverDnDpotri
    (cusolverH,HIPBLAS_FILL_MODE_LOWER,n,d_A,n,d_work,
     work_size,devInfo);
  
  if (cusolver_status != HIPSOLVER_STATUS_SUCCESS) {
    hipFree(d_work);
    hipFree(devInfo);
    hipsolverDnDestroy(cusolverH);
    hipFree(d_A);
    return 8;
  }
  
  // Copy result back to host
  A_inv.resize(n*n);
  cudaStat=hipMemcpy(A_inv.data(),d_A,n*n*sizeof(double),
                      hipMemcpyDeviceToHost);
  if (cudaStat != hipSuccess) {
    return 9;
  }
  
  // Symmetrize the result (only lower triangle is filled)
  for (int i=0;i<n;++i) {
    for (int j=i+1;j<n;++j) {
      A_inv[i*n+j]=A_inv[j*n+i];
    }
  }
  
  // Clean up
  hipFree(d_A);
  hipFree(d_work);
  hipFree(devInfo);
  hipsolverDnDestroy(cusolverH);
  
  return 0;
}
  
double matrix_invert_det_cholesky_cuda::det
(size_t n, const std::vector<double> &A) {

  // Allocate device memory
  double *d_A=0;
  hipError_t cudaStat=hipMalloc((void**)&d_A,n*n*sizeof(double));
  if (cudaStat != hipSuccess) {
    return 1;
  }
  
  // Copy data to device
  cudaStat=hipMemcpy(d_A,A.data(),n*n*sizeof(double),
                      hipMemcpyHostToDevice);
  if (cudaStat != hipSuccess) {
    hipFree(d_A);
    return 2;
  }
  
  // Create cuSolver handle
  hipsolverHandle_t cusolverH=0;
  hipsolverStatus_t cusolver_status=hipsolverDnCreate(&cusolverH);
  if (cusolver_status != HIPSOLVER_STATUS_SUCCESS) {
    hipFree(d_A);
    return 3;
  }
  
  // Get buffer size for potrf (Cholesky)
  int work_size=0;
  cusolver_status=hipsolverDnDpotrf_bufferSize
    (cusolverH,HIPBLAS_FILL_MODE_LOWER,n,d_A,n,&work_size);
  
  if (cusolver_status != HIPSOLVER_STATUS_SUCCESS) {
    hipsolverDnDestroy(cusolverH);
    hipFree(d_A);
    return 4;
  }
  
  // Allocate workspace and devInfo
  double *d_work=0;
  int *devInfo=0;
  cudaStat=hipMalloc((void**)&d_work,work_size*sizeof(double));
  if (cudaStat != hipSuccess) {
    hipsolverDnDestroy(cusolverH);
    hipFree(d_A);
    return 5;
  }
  
  cudaStat=hipMalloc((void**)&devInfo,sizeof(int));
  if (cudaStat != hipSuccess) {
    hipFree(d_work);
    hipsolverDnDestroy(cusolverH);
    hipFree(d_A);
    return 6;
  }
  
  // Cholesky decomposition (A=L*L^T)
  cusolver_status=hipsolverDnDpotrf
    (cusolverH,HIPBLAS_FILL_MODE_LOWER,n,d_A,n,d_work,
     work_size,devInfo);
  
  if (cusolver_status != HIPSOLVER_STATUS_SUCCESS) {
    hipFree(d_work);
    hipFree(devInfo);
    hipsolverDnDestroy(cusolverH);
    hipFree(d_A);
    return 7;
  }

  // Copy Cholesky decomposition back to host to compute
  // determinant
  std::vector<double> chol(n*n);
  cudaStat=hipMemcpy(chol.data(),d_A,n*n*sizeof(double),
                      hipMemcpyDeviceToHost);

  // Compute determinant
  double sqrt_det=1.0;
  for(size_t i=0;i<n;i++) sqrt_det*=chol[i*n+i];
  double A_det=sqrt_det*sqrt_det;
  
  hipFree(d_work);
  hipFree(devInfo);
  hipsolverDnDestroy(cusolverH);
  hipFree(d_A);

  return A_det;
}
  
int matrix_invert_det_cholesky_cuda::invert_inplace
(size_t n, std::vector<double> &A) {
  return invert(n,A,A);
}


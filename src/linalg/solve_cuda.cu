/*
  ───────────────────────────────────────────────────────────────────

  Copyright (C) 2025, Andrew W. Steiner

  This file is part of O2scl.

  O2scl is free software; you can redistribute it and/or modify
  it under the terms of the GNU General Public License as published by
  the Free Software Foundation; either version 3 of the License, or
  (at your option) any later version.

  O2scl is distributed in the hope that it will be useful,
  but WITHOUT ANY WARRANTY; without even the implied warranty of
  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
  GNU General Public License for more details.

  You should have received a copy of the GNU General Public License
  along with O2scl. If not, see <http://www.gnu.org/licenses/>.

  ───────────────────────────────────────────────────────────────────
*/
#include "solve_cuda.h"

#include <iostream>

#include <hip/hip_runtime.h>
#include <hipsolver.h>
#include <hipblas.h>

using namespace std;
using namespace o2scl_linalg;

int linear_solver_LU_cuda_base::solve_base
(int n, const std::vector<double> &A, const std::vector<double> &b,
 std::vector<double> &x) {
  
  double *d_A;
  double *d_b;
  int *d_pivots;
  int *d_info;
  double *d_work;
  int work_size=0;

  x.resize(n);
  
  hipsolverHandle_t cusolverH;
  if (hipsolverDnCreate(&cusolverH)!=HIPSOLVER_STATUS_SUCCESS) {
    return 1;
  }

  if (hipMalloc(&d_A,n*n*sizeof(double))!=hipSuccess) {
    hipsolverDnDestroy(cusolverH);
    return 5;
  }
  if (hipMalloc(&d_b,n*sizeof(double))!=hipSuccess) {
    hipFree(d_A);
    hipsolverDnDestroy(cusolverH);
    return 6;
  }
  if (hipMalloc(&d_pivots,n*sizeof(int))!=hipSuccess) {
    hipFree(d_A);
    hipFree(d_b);
    hipsolverDnDestroy(cusolverH);
    return 7;
  }
  if (hipMalloc(&d_info,sizeof(int))!=hipSuccess) {
    hipFree(d_A);
    hipFree(d_b);
    hipFree(d_pivots);
    hipsolverDnDestroy(cusolverH);
    return 8;
  }

  if (hipMemcpy(d_A,&(A[0]),n*n*sizeof(double),
                 hipMemcpyHostToDevice)!=hipSuccess) {
    hipFree(d_A);
    hipFree(d_b);
    hipFree(d_pivots);
    hipFree(d_info);
    hipsolverDnDestroy(cusolverH);
    return 9;
  }
  if (hipMemcpy(d_b,&(b[0]),n*sizeof(double),hipMemcpyHostToDevice)!=
      hipSuccess) {
    hipFree(d_A);
    hipFree(d_b);
    hipFree(d_pivots);
    hipFree(d_info);
    hipsolverDnDestroy(cusolverH);
    return 10;
  }

  // Query working space
  if (hipsolverDnDgetrf_bufferSize(cusolverH,n,n,d_A,n,&work_size)!=
      HIPSOLVER_STATUS_SUCCESS) {
    hipFree(d_A);
    hipFree(d_b);
    hipFree(d_pivots);
    hipFree(d_info);
    hipsolverDnDestroy(cusolverH);
    return 2;
  }
  if (hipMalloc(&d_work,work_size*sizeof(double))!=hipSuccess) {
    hipFree(d_A);
    hipFree(d_b);
    hipFree(d_pivots);
    hipFree(d_info);
    hipsolverDnDestroy(cusolverH);
    return 11;
  }

  // LU decomposition (with partial pivoting)
  if (hipsolverDnDgetrf(cusolverH,n,n,d_A,n,d_work,d_pivots,d_info)!=
      HIPSOLVER_STATUS_SUCCESS) {
    hipFree(d_A);
    hipFree(d_b);
    hipFree(d_pivots);
    hipFree(d_info);
    hipFree(d_work);
    hipsolverDnDestroy(cusolverH);
    return 3;
  }

  // Solve using LU
  if (hipsolverDnDgetrs(cusolverH,HIPBLAS_OP_N,n,1,d_A,n,d_pivots,
                       d_b,n,d_info)!=HIPSOLVER_STATUS_SUCCESS) {
    hipFree(d_A);
    hipFree(d_b);
    hipFree(d_pivots);
    hipFree(d_info);
    hipFree(d_work);
    hipsolverDnDestroy(cusolverH);
    return 4;
  }

  // Copy result
  if (hipMemcpy(&(x[0]),d_b,n*sizeof(double),
                 hipMemcpyDeviceToHost)!=hipSuccess) {
    hipFree(d_A);
    hipFree(d_b);
    hipFree(d_pivots);
    hipFree(d_info);
    hipFree(d_work);
    hipsolverDnDestroy(cusolverH);
    return 12;
  }

  // Cleanup
  hipFree(d_A);
  hipFree(d_b);
  hipFree(d_pivots);
  hipFree(d_info);
  hipFree(d_work);
  hipsolverDnDestroy(cusolverH);

  return 0;
}

int main(void) {

  vector<double> A={1,0,0,0,2,0,0,0,3};
  vector<double> b={4,5,6}, x(3);

  linear_solver_LU_cuda lslc;
  int ret=lslc.solve(3,A,b,x);
  
  cout << ret << " " << x[0] << " " << x[1] << " " << x[2] << endl;

  return 0;
}

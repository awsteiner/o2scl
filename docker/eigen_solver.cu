#include <iostream>
#include <hipsolver.h>
#include <hip/hip_runtime.h>

void checkCuda(hipError_t result) {
    if (result != hipSuccess) {
        std::cerr << "CUDA Runtime Error: " << hipGetErrorString(result) << std::endl;
        exit(EXIT_FAILURE);
    }
}

void checkCusolver(hipsolverStatus_t result) {
    if (result != HIPSOLVER_STATUS_SUCCESS) {
        std::cerr << "cuSOLVER Error: " << result << std::endl;
        exit(EXIT_FAILURE);
    }
}

int main() {
    hipsolverHandle_t cusolverH;
    checkCusolver(hipsolverDnCreate(&cusolverH));

    const int N = 3;  // Matrix size
    double A[N * N] = {3.0, 2.0, 4.0, 
                       2.0, 0.0, 2.0, 
                       4.0, 2.0, 3.0};  // Symmetric matrix

    double *d_A, *d_W;
    int *d_info, Lwork;
    double *d_work;

    checkCuda(hipMalloc((void**)&d_A, N * N * sizeof(double)));
    checkCuda(hipMalloc((void**)&d_W, N * sizeof(double)));
    checkCuda(hipMalloc((void**)&d_info, sizeof(int)));

    checkCuda(hipMemcpy(d_A, A, N * N * sizeof(double), hipMemcpyHostToDevice));

    // Query workspace size
    checkCusolver(hipsolverDnDsyevd_bufferSize(cusolverH, HIPSOLVER_EIG_MODE_VECTOR, HIPBLAS_FILL_MODE_LOWER, N, d_A, N, d_W, &Lwork));

    checkCuda(hipMalloc((void**)&d_work, Lwork * sizeof(double)));

    // Compute eigenvalues and eigenvectors
    checkCusolver(hipsolverDnDsyevd(cusolverH, HIPSOLVER_EIG_MODE_VECTOR, HIPBLAS_FILL_MODE_LOWER, N, d_A, N, d_W, d_work, Lwork, d_info));

    // Copy results back to host
    double W[N];
    checkCuda(hipMemcpy(W, d_W, N * sizeof(double), hipMemcpyDeviceToHost));

    // Print eigenvalues
    std::cout << "Eigenvalues: ";
    for (int i = 0; i < N; i++) {
        std::cout << W[i] << " ";
    }
    std::cout << std::endl;

    // Clean up
    checkCuda(hipFree(d_A));
    checkCuda(hipFree(d_W));
    checkCuda(hipFree(d_info));
    checkCuda(hipFree(d_work));
    checkCusolver(hipsolverDnDestroy(cusolverH));

    return 0;
}
